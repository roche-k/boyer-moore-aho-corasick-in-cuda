#include "hip/hip_runtime.h"
#include "project-header-bits.h"

string patterns[]={"ATC","GTG","GTC","ATG","CAA","ATT"};

texture<int, hipTextureType2D> tex_go_to_function;
texture<unsigned int, hipTextureType1D> tex_failure_function;
texture<unsigned int, hipTextureType1D> tex_output_function;

__global__ void shared_kernel1 ( unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks, int sharedMemSize ) {

	//int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int r, s;
	
	int i, j, column;
	
	int charactersPerThread = sharedMemSize / blockDim.x;
	
	int startThread = charactersPerThread * threadIdx.x;
	int stopThread = startThread + charactersPerThread + m - 1;

	//Define space in shared memory
	extern __shared__ unsigned char s_array[];
	
	for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += numBlocks * sharedMemSize ) {
	
		/*if ( threadIdx.x == 0 )
			for ( i = globalMemIndex, j = 0; ( j < sharedMemSize + m - 1 && i < n ); i++, j++ )
				s_array[j] = d_text[i];
		*/
		
		for ( i = globalMemIndex + threadIdx.x, j = 0 + threadIdx.x; ( j < sharedMemSize + m - 1 && i < n ); i+=blockDim.x, j+=blockDim.x )
			s_array[j] = d_text[i];
			
		__syncthreads();
		
		r = 0;
		
		for ( column = startThread; ( column < stopThread && globalMemIndex + column < n ); column++ ) {
		
			while ( ( s = tex2D ( tex_go_to_function, s_array[column], r ) ) == -1 )
				r = tex1Dfetch ( tex_failure_function, r );
			r = s;
			
			d_out[globalMemIndex + column] += tex1Dfetch ( tex_output_function, r );
		}
		
		__syncthreads();
	}
}


void shared1 ( int m, unsigned char *text, int n, int p_size, int alphabet, int *go_to_function, unsigned int *failure_function, unsigned int *output_function ) {

	//Pointer for device memory
	int *d_go_to_function;
	unsigned int *d_failure_function, *d_output_function, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 24, numThreadsPerBlock = 1024, sharedMemSize = 16384;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( n * sizeof ( unsigned int ) );
	memset ( h_out, 0, n * sizeof ( unsigned int ) );
	
	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_failure_function, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_output_function, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, n * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_go_to_function, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_failure_function, failure_function, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_output_function, output_function, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, n * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	
	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_go_to_function, pitch, go_to_function, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	checkCudaErrors ( hipBindTexture2D ( 0, tex_go_to_function, d_go_to_function, desc, alphabet, m * p_size + 1, pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_failure_function, d_failure_function, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_output_function, d_output_function, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	shared_kernel1<<<dimGrid, dimBlock, sharedMemSize + m - 1>>>( d_text, d_out, m, n, p_size, alphabet, numBlocks, sharedMemSize );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, n * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	   
  	//Look at the results
  	int i, matches = 0;
  	vector < vector<int> > indices;
  	for(int i=0; i<D; i++)
  	{
  		vector <int> row;
  		indices.push_back(row);
  	}
  	
  	for ( i = 0; i < n; i++ )
  	{
  		int count = 0;
  		if(h_out[i] == 0) continue;
  		for (int j = 0; j < D; ++j)
                {
                    if (h_out[i] & (1 << j))
                    {
                       // cout << "Word " << arr[j] << " appears from "
                        //    << i - arr[j].size() + 1 << " to " << i << endl;
                        //cout<<h_out[i]<<" ";
                        indices[j].push_back(i - M + 1);
                        count++;
                    }
                }
  		matches += count;
  	}
  	
  	
	printf ("Shared Memory Kernel 1 matches \t%i\t time \t%fms\n", matches, time);

	for(int i=0; i<D; i++){
		ofstream outputfile(patterns[i] + ".txt");
		cout<<indices[i].size()<<" ";
		for(int j=0;j<indices[i].size();j++)
        	outputfile<<indices[i][j]<<"\n";
	}

	printf("Pattern occurences written to individual files.");
	
	hipUnbindTexture ( tex_go_to_function );
	hipUnbindTexture ( tex_failure_function );
	hipUnbindTexture ( tex_output_function );
	
	//Free host and device memory
	free ( h_out );

	hipFree ( d_text );
	hipFree ( d_go_to_function );
	hipFree ( d_failure_function );
	hipFree ( d_output_function );
	hipFree ( d_out );
}


int main(){

	int k = sizeof(patterns)/sizeof(patterns[0]);
	string text;

	std::ifstream t("data.txt");
	std::stringstream buffer;
	buffer << t.rdbuf();	
	text = buffer.str();
	unsigned char *charText = (unsigned char*)text.c_str();
	
	buildMatchingMachine(patterns, k);

	int *goToTable = (int*)malloc(sizeof(int)*MAXC*MAXS);
	for(int i=0;i<MAXS;i++)
		for(int j=0;j<MAXC;j++)
			goToTable[i*MAXC+j] = g[i][j];

	shared1(M,charText,text.size(),D,26,goToTable,f,out);
	return 0;
}

