#include "hip/hip_runtime.h"
#include "project-header.h"

texture<int, hipTextureType2D> tex_go_to_state;
texture<unsigned int, hipTextureType1D> tex_failure_state;
texture<unsigned int, hipTextureType1D> tex_output_state;

__global__ void shared_kernel2 ( unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int num_blocks, int sharedMemSize ) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int r, s;
    int i, j, column, matches = 0;
    int chars_per_thread = sharedMemSize / blockDim.x;
    int start_thread = chars_per_thread * threadIdx.x;
    int stop_thread = start_thread + chars_per_thread + m - 1;

    //Define space in shared memory
    extern __shared__ unsigned char s_array[];
    
    //cast data to uint4
    uint4 *uint4_text = reinterpret_cast < uint4 * > ( d_text );
    uint4 uint4_var;
    
    //recast data to uchar4
    uchar4 c0, c4, c8, c12;
    
    for ( int globalMemIndex = blockIdx.x * sharedMemSize; globalMemIndex < n; globalMemIndex += num_blocks * sharedMemSize ) {
    
        for ( i = globalMemIndex/16 + threadIdx.x, j = 0 + threadIdx.x; j < sharedMemSize / 16 && i < n / 16; i+=blockDim.x, j+=blockDim.x ) {
            
            uint4_var = uint4_text[i];
            
            //recast data back to char after the memory transaction
            c0 = *reinterpret_cast<uchar4 *> ( &uint4_var.x );
            c4 = *reinterpret_cast<uchar4 *> ( &uint4_var.y );
            c8 = *reinterpret_cast<uchar4 *> ( &uint4_var.z );
            c12 = *reinterpret_cast<uchar4 *> ( &uint4_var.w );

                        s_array[j * 16 + 0] = c0.x;
                        s_array[j * 16 + 1] = c0.y;
                        s_array[j * 16 + 2] = c0.z;
                        s_array[j * 16 + 3] = c0.w;
                        
                        s_array[j * 16 + 4] = c4.x;
                        s_array[j * 16 + 5] = c4.y;
                        s_array[j * 16 + 6] = c4.z;
                        s_array[j * 16 + 7] = c4.w;
                        
                        s_array[j * 16 + 8] = c8.x;
                        s_array[j * 16 + 9] = c8.y;
                        s_array[j * 16 + 10] = c8.z;
                        s_array[j * 16 + 11] = c8.w;
                        
                        s_array[j * 16 + 12] = c12.x;
                        s_array[j * 16 + 13] = c12.y;
                        s_array[j * 16 + 14] = c12.z;
                        s_array[j * 16 + 15] = c12.w;
        }

        //Add m - 1 redundant characters at the end of the shared memory
        if ( threadIdx.x < m - 1 )
            s_array[sharedMemSize + threadIdx.x] = d_text[globalMemIndex + sharedMemSize + threadIdx.x];
            
        __syncthreads();
        
        r = 0;
        
        for ( column = start_thread; ( column < stop_thread && globalMemIndex + column < n ); column++ ) {
        
            while ( ( s = tex2D ( tex_go_to_state, s_array[column], r ) ) == -1 )
                r = tex1Dfetch ( tex_failure_state, r );
            r = s;
            
            matches += tex1Dfetch ( tex_output_state, r );
        }
        
        __syncthreads();
    }
    
    d_out[idx] = matches;
}

void shared2(int m, unsigned char *text, int n, int p_size, int alphabet, int *go_to_state, unsigned int *failure_state, unsigned int *output_state ) {

    //Pointer for device memory
    int *d_go_to_state;
    unsigned int *d_failure_state, *d_output_state, *d_out;
    
    unsigned char *d_text;

    size_t pitch;
    
    int num_blocks = 24, num_threads_per_block = 1024, sharedMemSize = 16384;
    dim3 dimGrid ( num_blocks );
    dim3 dimBlock ( num_threads_per_block );
    
    if ( n < num_blocks * num_threads_per_block * m ) {
        printf("The text size is too small\n");
        exit(1);
    }
    
    //Allocate host memory for results array
    unsigned int *h_out = ( unsigned int * ) malloc ( num_blocks * num_threads_per_block * sizeof ( unsigned int ) );
    memset ( h_out, 0, num_blocks * num_threads_per_block * sizeof ( unsigned int ) );
    
    //Allocate 1D device memory
    checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
    checkCudaErrors ( hipMalloc ( ( void** ) &d_failure_state, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
    checkCudaErrors ( hipMalloc ( ( void** ) &d_output_state, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
    checkCudaErrors ( hipMalloc ( ( void** ) &d_out, num_blocks * num_threads_per_block * sizeof ( unsigned int ) ) );
    
    //Allocate 2D device memory
    checkCudaErrors ( hipMallocPitch ( &d_go_to_state, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
    
    //Copy 1D host memory to device
    checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
    checkCudaErrors ( hipMemcpy ( d_failure_state, failure_state, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
    checkCudaErrors ( hipMemcpy ( d_output_state, output_state, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
    checkCudaErrors ( hipMemcpy ( d_out, h_out, num_blocks * num_threads_per_block * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
    
    //Copy 2D host memory to device
    checkCudaErrors ( hipMemcpy2D ( d_go_to_state, pitch, go_to_state, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
    
    //Bind the preprocessing tables to the texture cache
    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    checkCudaErrors ( hipBindTexture2D ( 0, tex_go_to_state, d_go_to_state, desc, alphabet, m * p_size + 1, pitch ) );
    checkCudaErrors ( hipBindTexture ( 0, tex_failure_state, d_failure_state, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
    checkCudaErrors ( hipBindTexture ( 0, tex_output_state, d_output_state, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
    
    //Create timer
    hipEvent_t start, stop;

    float time;

    //Create the timer events
    hipEventCreate ( &start );
    hipEventCreate ( &stop );
    
    //Start the event clock	
    hipEventRecord ( start, 0 );
    
    //Executing kernel in the device
    shared_kernel2<<<dimGrid, dimBlock, sharedMemSize + 16 * ( ( m - 1 ) / 16 + 1 )>>>( d_text, d_out, m, n, p_size, alphabet, num_blocks, sharedMemSize );
    checkCUDAError("kernel invocation");
    
    hipEventRecord ( stop, 0 );

    hipEventSynchronize ( stop );
    
    hipEventElapsedTime ( &time, start, stop );
    
    hipEventDestroy ( start );
    hipEventDestroy ( stop );

    //Get back the results from the device
    hipMemcpy ( h_out, d_out, num_blocks * num_threads_per_block * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
       
      //Look at the results
      int i, matches = 0;
      
      for ( i = 0; i < num_blocks * num_threads_per_block; i++ )
          matches += h_out[i];
      
    printf ("Kernel 5 matches \t%i\t time \t%fms\n", matches, time);
            
    hipUnbindTexture ( tex_go_to_state );
    hipUnbindTexture ( tex_failure_state );
    hipUnbindTexture ( tex_output_state );
    
    //Free host and device memory
    free ( h_out );

    hipFree ( d_text );
    hipFree ( d_go_to_state );
    hipFree ( d_failure_state );
    hipFree ( d_output_state );
    hipFree ( d_out );
}


int main(){

    string patterns[]={"ATC","GTG","GTC","ATG","CAA","ATT"};
    int k = sizeof( patterns )/sizeof( patterns[0] );
    string text;

    ifstream t( "data.txt" );
    stringstream buffer;
    buffer << t.rdbuf();    
    text = buffer.str();
    unsigned char *charText = ( unsigned char* )text.c_str();
    
    buildMatchingMachine(patterns, k);

    int *go_to_table = (int*)malloc( sizeof(int)*MAXC*MAXS );
    
    for(int i=0;i<MAXS;i++)
        for(int j=0;j<MAXC;j++)
            go_to_table[i*MAXC+j] = g[i][j];

    shared2( M, charText, text.size(), D, 256, go_to_table, f, out );
    return 0;
}

